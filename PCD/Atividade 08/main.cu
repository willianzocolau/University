
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TAM 1.0
#define DX 0.00001
#define DT 0.000001
#define T  0.01
#define KAPPA 0.000045
#define N (long int)(TAM/DX)
#define CONSTANT KAPPA * DT / (DX*DX)

#define THREADS_PER_BLOCK 512
#define BLOCK_SIZE 2

__global__ void calculator(double *a, double *b);

void change_context(double **a, double **b);

int main() {
  long int i, maxloc, size = (N + 1) * sizeof(double);;
  double *d_a, *d_b, x = 0, t = 0.;
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  double *a = (double *) malloc(size);
  double *b = (double *) malloc(size);

  printf("Início: qtde = %ld, dt = %g, dx = %g, dx² = %g, kappa = %f, const = %f\n", (N + 1), DT, DX, DX * DX, KAPPA, KAPPA * DT / (DX * DX));
  printf("Iterações previstas: %g\n", T / DT);

  for (i = 0; i < N + 1; i++) {
    if (x <= 0.5) {
      a[i] = 200 * x;
    } else {
      a[i] = 200 * (1. - x);
    }
    x += DX;
  }

  printf("dx = %g, x = %g, (x-dx) = %g\n", DX, x, x - DX);
  printf("prev[0,1] = %g, %g\n", a[0], a[1]);
  printf("prev[n-1,n] = %g, %g\n", a[N - 1], a[N]);

  while (t < T) {
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    calculator<<< ((N - 1) + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b);
    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    b[0] = b[N] = 0.;
    change_context(&a, &b);
    t += DT;
  }

  maxloc = 0;
  for (i = 1; i < N + 1; i++) if (b[i] > b[maxloc]) maxloc = i;
  printf("Maior valor u[%ld] = %g\n", maxloc, b[maxloc]);
  hipFree(d_a);
  hipFree(d_b);
  return 0;
}

__global__ void calculator(double *a, double *b) {
  int i = (blockIdx.x * blockDim.x + threadIdx.x) + 1;
  b[i] = a[i] + CONSTANT * (a[i - 1] - 2 * a[i] + a[i + 1]);
}

void change_context(double **a, double **b) {
  double *temp;
  temp = *b;
  *b = *a;
  *a = temp;
}
